#include "CudaScene.cuh"

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

// project
#include "Relation.hpp"


extern "C"
{


  namespace set
  {

  namespace
  {

  thrust::device_vector< Relation0 > *pGeomVec;

  }



  ///
  /// \brief cuda_initScene
  ///
  void
  cuda_initScene( )
  {
    pGeomVec = new thrust::device_vector< Relation0 >( );
  }



  ///
  /// \brief cuda_addRelation
  /// \param x
  /// \param y
  /// \param z
  ///
  void
  cuda_addRelation(
                   float x,
                   float y,
                   float z
                   )
  {
    pGeomVec->push_back( Relation0 ( x, y, z ) );
  }



  ///
  /// \brief cuda_clearScene
  ///
  void
  cuda_clearScene( )
  {
    pGeomVec->clear( );
    pGeomVec->shrink_to_fit( );
    delete pGeomVec;
  }



  } // namespace set


} // extern "C"
